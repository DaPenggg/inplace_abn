#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_fp16.h>

#include <vector>

#include "common.h"
#include "inplace_abn.h"

// Operations for reduce
struct SumOpH {
  __device__ SumOpH(const half *t, int c, int s)
      : tensor(t), chn(c), sp(s) {}
  __device__ __forceinline__ float operator()(int batch, int plane, int n) {
    return __half2float(tensor[(batch * chn + plane) * sp + n]);
  }
  const half *tensor;
  const int chn;
  const int sp;
};

struct VarOpH {
  __device__ VarOpH(float m, const half *t, int c, int s)
      : mean(m), tensor(t), chn(c), sp(s) {}
  __device__ __forceinline__ float operator()(int batch, int plane, int n) {
    const auto t = __half2float(tensor[(batch * chn + plane) * sp + n]);
    return (t - mean) * (t - mean);
  }
  const float mean;
  const half *tensor;
  const int chn;
  const int sp;
};


struct GradOp {
  __device__ GradOp(float _weight, float _bias, const half *_z, const half *_dz, int c, int s)
      : weight(_weight), bias(_bias), z(_z), dz(_dz), chn(c), sp(s) {}
  __device__ __forceinline__ Pair<float> operator()(int batch, int plane, int n) {
    float _y = (__half2float(z[(batch * chn + plane) * sp + n]) - bias) / weight;
    float _dz = __half2float(dz[(batch * chn + plane) * sp + n]);
    return Pair<float>(_dz, _y * _dz);
  }
  const float weight;
  const float bias;
  const half *z;
  const half *dz;
  const int chn;
  const int sp;
};

/***********
 * mean_var
 ***********/

__global__ void mean_var_kernel_h(const half *x, float *mean, float *var, int num, int chn, int sp) {
  int plane = blockIdx.x;
  float norm = 1.f / static_cast<float>(num * sp);

  float _mean = reduce<float, SumOpH>(SumOpH(x, chn, sp), plane, num, sp) * norm;
  __syncthreads();
  float _var = reduce<float, VarOpH>(VarOpH(_mean, x, chn, sp), plane, num, sp) * norm;

  if (threadIdx.x == 0) {
    mean[plane] = _mean;
    var[plane] = _var;
  }
}

std::vector<at::Tensor> mean_var_cuda_h(at::Tensor x) {
  CHECK_INPUT(x);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(x, num, chn, sp);

  // Prepare output tensors
  auto mean = at::empty(x.type().toScalarType(at::kFloat), {chn});
  auto var = at::empty(x.type().toScalarType(at::kFloat), {chn});

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  mean_var_kernel_h<<<blocks, threads>>>(
      reinterpret_cast<half*>(x.data<at::Half>()),
      mean.data<float>(),
      var.data<float>(),
      num, chn, sp);

  return {mean, var};
}

/**********
 * forward
 **********/

__global__ void forward_kernel_h(half *x, const float *mean, const float *var, const float *weight, const float *bias,
                                 bool affine, float eps, int num, int chn, int sp) {
  int plane = blockIdx.x;

  const float _mean = mean[plane];
  const float _var = var[plane];
  const float _weight = affine ? abs(weight[plane]) + eps : 1.f;
  const float _bias = affine ? bias[plane] : 0.f;

  const float mul = rsqrt(_var + eps) * _weight;

  for (int batch = 0; batch < num; ++batch) {
    for (int n = threadIdx.x; n < sp; n += blockDim.x) {
      half *x_ptr = x + (batch * chn + plane) * sp + n;
      float _x = __half2float(*x_ptr);
      float _y = (_x - _mean) * mul + _bias;

      *x_ptr = __float2half(_y);
    }
  }
}

at::Tensor forward_cuda_h(at::Tensor x, at::Tensor mean, at::Tensor var, at::Tensor weight, at::Tensor bias,
                        bool affine, float eps) {
  CHECK_INPUT(x);
  CHECK_INPUT(mean);
  CHECK_INPUT(var);
  CHECK_INPUT(weight);
  CHECK_INPUT(bias);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(x, num, chn, sp);

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  forward_kernel_h<<<blocks, threads>>>(
      reinterpret_cast<half*>(x.data<at::Half>()),
      mean.data<float>(),
      var.data<float>(),
      weight.data<float>(),
      bias.data<float>(),
      affine, eps, num, chn, sp);

  return x;
}

__global__ void edz_eydz_kernel_h(const half *z, const half *dz, const float *weight, const float *bias,
                                float *edz, float *eydz, bool affine, float eps, int num, int chn, int sp) {
  int plane = blockIdx.x;

  float _weight = affine ? abs(weight[plane]) + eps : 1.f;
  float _bias = affine ? bias[plane] : 0.f;

  Pair<float> res = reduce<Pair<float>, GradOp>(GradOp(_weight, _bias, z, dz, chn, sp), plane, num, sp);
  __syncthreads();

  if (threadIdx.x == 0) {
    edz[plane] = res.v1;
    eydz[plane] = res.v2;
  }
}

std::vector<at::Tensor> edz_eydz_cuda_h(at::Tensor z, at::Tensor dz, at::Tensor weight, at::Tensor bias,
                                      bool affine, float eps) {
  CHECK_INPUT(z);
  CHECK_INPUT(dz);
  CHECK_INPUT(weight);
  CHECK_INPUT(bias);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(z, num, chn, sp);

  auto edz = at::empty(z.type().toScalarType(at::kFloat), {chn});
  auto eydz = at::empty(z.type().toScalarType(at::kFloat), {chn});

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  edz_eydz_kernel_h<<<blocks, threads>>>(
        reinterpret_cast<half*>(z.data<at::Half>()),
        reinterpret_cast<half*>(dz.data<at::Half>()),
        weight.data<float>(),
        bias.data<float>(),
        edz.data<float>(),
        eydz.data<float>(),
        affine, eps, num, chn, sp);
 
  return {edz, eydz};
}

__global__ void backward_kernel_h(const half *z, const half *dz, const float *var, const float *weight, const float *bias, const float *edz,
                                const float *eydz, half *dx, float *dweight, float *dbias,
                                bool affine, float eps, int num, int chn, int sp) {
  int plane = blockIdx.x;

  float _weight = affine ? abs(weight[plane]) + eps : 1.f;
  float _bias = affine ? bias[plane] : 0.f;
  float _var = var[plane];
  float _edz = edz[plane];
  float _eydz = eydz[plane];

  float _mul = _weight * rsqrt(_var + eps);
  float count = float(num * sp);

  for (int batch = 0; batch < num; ++batch) {
    for (int n = threadIdx.x; n < sp; n += blockDim.x) {
      float _dz = __half2float(dz[(batch * chn + plane) * sp + n]);
      float _y = (__half2float(z[(batch * chn + plane) * sp + n]) - _bias) / _weight;

      dx[(batch * chn + plane) * sp + n] = __float2half((_dz - _edz / count - _y * _eydz / count) * _mul);
    }
  }

  if (threadIdx.x == 0) {
    if (affine) {
      dweight[plane] = weight[plane] > 0 ? _eydz : -_eydz;
      dbias[plane] = _edz;
    }
  }
}

std::vector<at::Tensor> backward_cuda_h(at::Tensor z, at::Tensor dz, at::Tensor var, at::Tensor weight, at::Tensor bias,
                                      at::Tensor edz, at::Tensor eydz, bool affine, float eps) {
  CHECK_INPUT(z);
  CHECK_INPUT(dz);
  CHECK_INPUT(var);
  CHECK_INPUT(weight);
  CHECK_INPUT(bias);
  CHECK_INPUT(edz);
  CHECK_INPUT(eydz);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(z, num, chn, sp);

  auto dx = at::zeros_like(z);
  auto dweight = at::zeros_like(weight);
  auto dbias = at::zeros_like(bias);

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  backward_kernel_h<<<blocks, threads>>>(
        reinterpret_cast<half*>(z.data<at::Half>()),
        reinterpret_cast<half*>(dz.data<at::Half>()),
        var.data<float>(),
        weight.data<float>(),
        bias.data<float>(),
        edz.data<float>(),
        eydz.data<float>(),
        reinterpret_cast<half*>(dx.data<at::Half>()),
        dweight.data<float>(),
        dbias.data<float>(),
        affine, eps, num, chn, sp);

  return {dx, dweight, dbias};
}

__global__ void leaky_relu_backward_impl_h(half *z, half *dz, float slope, int64_t count) {
   for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count;  i += blockDim.x * gridDim.x){
       float _z = __half2float(z[i]);
       if(_z<0){
	       dz[i]=__float2half(__half2float(dz[i])*slope);
	       z[i]=__float2half(_z/slope);
       }
   }
}

void leaky_relu_backward_cuda_h(at::Tensor z, at::Tensor dz, float slope) {
  CHECK_INPUT(z);
  CHECK_INPUT(dz);

  int64_t count = z.numel();
  dim3 threads(512);
  dim3 blocks = (count + threads.x - 1) / threads.x;

  leaky_relu_backward_impl_h<<<blocks,threads>>>(
        reinterpret_cast<half*>(z.data<at::Half>()),
        reinterpret_cast<half*>(dz.data<at::Half>()),
        slope, count
  );
}

